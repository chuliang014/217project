#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include "malloc.h"
#include "string.h" 
#include "parboil.h"

#include "file.h"
#include "computeQ.cu"

static void setupMemoryGPU(int num, int size, float*& dev_ptr, float*& host_ptr)
{
  hipMalloc ((void **) &dev_ptr, num * size);
  hipMemcpy (dev_ptr, host_ptr, num * size, hipMemcpyHostToDevice);
}

static void
cleanupMemoryGPU(int num, int size, float *& dev_ptr, float * host_ptr)
{
  hipMemcpy (host_ptr, dev_ptr, num * size, hipMemcpyDeviceToHost);
  hipFree(dev_ptr);
}

int main (int argc, char *argv[]) {
  int numX, numK;		/* Number of X and K values */
  int original_numK;		/* Number of K values in input file */
  float *kx, *ky, *kz;		/* K trajectory (3D vectors) */
  float *x, *y, *z;		/* X coordinates (3D vectors) */
  float *phiR, *phiI;		/* Phi values (complex) */
  float *phiMag;		/* Magnitude of Phi */
  float *Qr, *Qi;		/* Q signal (complex) */
  struct kValues* kVals;

  struct pb_Parameters *params;
  struct pb_TimerSet timers;

  pb_InitializeTimerSet(&timers);

  /* Read command line */
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
    {
      fprintf(stderr, "Expecting one input filename\n");
      exit(-1);
    }
  
  /* Read in data */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  inputData(params->inpFiles[0],
	    &original_numK, &numX,
	    &kx, &ky, &kz,
	    &x, &y, &z,
	    &phiR, &phiI);

/* Reduce the number of k-space samples if a number is given
   * on the command line */
  if (argc < 2)
    numK = original_numK;
  else
    {
	int inputK;
    char *end;
	inputK = strtol(argv[1], &end, 10);
	if (end == argv[1])
		{
		fprintf(stderr, "Expecting an integer parameter\n");
		exit(-1);
		}
	numK = MIN(inputK, original_numK);
    }

	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

	printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
         numX, original_numK, numK);

  /* Create CPU data structures */
  createDataStructsCPU(numK, numX, &phiMag, &Qr, &Qi);

/* GPU1 precompute PhiMag */ 
  {
    float *phiR_d, *phiI_d;
    float *phiMag_d;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    setupMemoryGPU(numK, sizeof(float), phiR_d, phiR);
    setupMemoryGPU(numK, sizeof(float), phiI_d, phiI);
    hipMalloc((void **)&phiMag_d, numK * sizeof(float));
	hipDeviceSynchronize();

    pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

    ComputePhiMagGPU(numK, phiR_d, phiI_d, phiMag_d);
    hipDeviceSynchronize();

	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    
	cleanupMemoryGPU(numK, sizeof(float), phiMag_d, phiMag);
    hipFree(phiR_d);
    hipFree(phiI_d);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  kVals = (struct kValues*)calloc(numK, sizeof (struct kValues));
  for (int k = 0; k < numK; k++) {
    kVals[k].Kx = kx[k];
    kVals[k].Ky = ky[k];
    kVals[k].Kz = kz[k];
    kVals[k].PhiMag = phiMag[k];
  }

free(phiMag);

  /* GPU2 computeQ */
  {
    float *x_d, *y_d, *z_d;
    float *Qr_d, *Qi_d;
	kValues* kVal_d;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    setupMemoryGPU(numX, sizeof(float), x_d, x);
    setupMemoryGPU(numX, sizeof(float), y_d, y);
    setupMemoryGPU(numX, sizeof(float), z_d, z);
    hipMalloc((void **)&kVal_d, numK * sizeof(struct kValues));
    hipMemcpy(kVal_d, kVals, numK * sizeof(struct kValues), hipMemcpyHostToDevice);
    hipMalloc((void **)&Qr_d, numX * sizeof(float));
    hipMemset((void *)Qr_d, 0, numX * sizeof(float));
    hipMalloc((void **)&Qi_d, numX * sizeof(float));
    hipMemset((void *)Qi_d, 0, numX * sizeof(float));
    hipDeviceSynchronize();

	pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

	computeQGPU(numK, numX, x_d, y_d, z_d, kVal_d, Qr_d, Qi_d);
    hipDeviceSynchronize();

	pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipFree(kVal_d);
    cleanupMemoryGPU(numX, sizeof(float), Qr_d, Qr);
    cleanupMemoryGPU(numX, sizeof(float), Qi_d, Qi);
  }

    pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

if (params->outFile)
    {
      /* Write Q to file */
     pb_SwitchToTimer(&timers, pb_TimerID_IO);
      outputData(params->outFile, Qr, Qi, numX);
     pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
    }

  free (kx);
  free (ky);
  free (kz);
  free (x);
  free (y);
  free (z);
  free (phiR);
  free (phiI);
  free (kVals);
  free (Qr);
  free (Qi);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);

  return 0;
}
