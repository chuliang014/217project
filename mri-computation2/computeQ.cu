
#include <hip/hip_runtime.h>
#include <cstdlib>

#define PI   3.1415926535897932384626433832795029f
#define PIx2 6.2831853071795864769252867665590058f

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#define K_ELEMS_PER_GRID  2048
#define K_PHIMAG_BLOCK_SIZE 512
#define K_Q_BLOCK_SIZE 256
#define K_Q_K_ELEMS_PER_GRID 1024

struct kValues {
    float Kx;
    float Ky;
    float Kz;
    float PhiMag;
  };

  __constant__ __device__ kValues kVal[K_Q_K_ELEMS_PER_GRID];

__global__ void ComputePhiMag_GPU(float* phiR, float* phiI, float* phiMag, int numK){
    int indexK  = blockIdx.x * K_PHIMAG_BLOCK_SIZE + threadIdx.x;
    if(indexK < numK){
        float real = phiR[indexK];
        float imag = phiI[indexK];
        phiMag[indexK] = real*real + imag*imag;
    }
}

__global__ void computeQ_GPU(int numK, int kGlobalIndex, float* x, float* y, float* z, 
    float* Qr, float* Qi){
        __shared__ float s_x,s_y,s_z,s_Qr,s_Qi;

        int xIndex = blockIdx.x * K_Q_BLOCK_SIZE + threadIdx.x;
           
        s_x = x[xIndex];
        s_y = y[xIndex];
        s_z = z[xIndex];
        s_Qr = Qr[xIndex];
        s_Qi = Qi[xIndex];
        int indexK = 0;
        //vincent if it's odd, initialize the first
        if(numK % 2){
            float expArg = PIx2 * (kVal[0].Kx * s_x +
                kVal[0].Ky * s_y +
                kVal[0].Kz * s_z);
                s_Qr += kVal[0].PhiMag * cosf(expArg);
                s_Qi += kVal[0].PhiMag * sinf(expArg);
                indexK++;
                kGlobalIndex++;
        }
        //vincent the rest are even and compute 2 time in every iteration
        for(; indexK < K_Q_K_ELEMS_PER_GRID && kGlobalIndex < numK; indexK+=2, kGlobalIndex+=2){
            float expArg = PIx2 * (kVal[indexK].Kx * s_x +
                kVal[indexK].Ky * s_y +
                kVal[indexK].Kz * s_z);
                
                s_Qr += kVal[indexK].PhiMag * cosf(expArg);
                s_Qi += kVal[indexK].PhiMag * sinf(expArg);

                int indexk_1 = indexK + 1;
                float expArg_1 = PIx2 * (kVal[indexk_1].Kx * s_x +
                    kVal[indexk_1].Ky * s_y +
                    kVal[indexk_1].Kz * s_z);

                    s_Qr += kVal[indexk_1].PhiMag * cosf(expArg_1);
                    s_Qi += kVal[indexk_1].PhiMag * sinf(expArg_1);
        }
        Qr[xIndex] = s_Qr;
        Qi[xIndex] = s_Qi;
}

void ComputePhiMagGPU(int numK, float* d_phiR, float* d_phiI, float* d_phiMag){
    int phiMag_block = (numK-1) / K_PHIMAG_BLOCK_SIZE + 1;
    dim3 DimPhiMagBlock(K_PHIMAG_BLOCK_SIZE,1);
    dim3 DimPhiMagGrid(phiMag_block,1);
    ComputePhiMag_GPU<<<DimPhiMagGrid, DimPhiMagBlock>>>(d_phiR, d_phiI, d_phiMag, numK);
}

void computeQGPU(int numK, int numX,float* d_x, float* d_y, float* d_z,
    kValues* kVals,float* d_Qr, float* d_Qi){
        int gridQ = (numK -1) / K_Q_K_ELEMS_PER_GRID + 1;
        int blockQ = (numX - 1) / K_Q_BLOCK_SIZE + 1;
        dim3 DimQBlock(K_Q_BLOCK_SIZE, 1);
        dim3 DimQGrid(blockQ,1);

        for(int i = 0; i < gridQ; i++){
            int QGridBase = i * K_Q_K_ELEMS_PER_GRID;
            kValues* kValsTile = kVals + QGridBase;
            int num = MIN(K_Q_K_ELEMS_PER_GRID, numK - QGridBase);
            hipMemcpyToSymbol(HIP_SYMBOL(kVal), kValsTile, num * sizeof(kValues), 0);
            computeQ_GPU<<<DimQGrid, DimQBlock>>>(numK,QGridBase,d_x,d_y,d_z,d_Qr,d_Qi);
        }
    }

    void createDataStructsCPU(int numK, int numX, float** phiMag,
        float** Qr, float** Qi){
            *phiMag = (float* ) malloc(numK * sizeof(float));
            *Qr = (float*) malloc(numX * sizeof (float));
            *Qi = (float*) malloc(numX * sizeof (float));
        }